#include"CUMemoryPool.cuh"
#include<Windows.h>

__device__ void* gGlobalMemory = nullptr;

CUMemoryPool::CUMemoryPool(unsigned long long initialSize)
	: mPageSize(LowerUnit(4)), mOffset(0)
	// mPageSize == 4KB
{
	if (initialSize == 0)
	{
		initialSize = 2147483648;
	}

	hipError_t error = hipMalloc(reinterpret_cast<void**>(gGlobalMemory), initialSize >> 1);
	if (error != NULL)
	{
		initialSize = initialSize >> 1;

		error = hipMalloc(reinterpret_cast<void**>(gGlobalMemory), initialSize >> 1);
	
		if (error != NULL)
		{
#ifdef _DEBUG
			std::cout << "Insufficient Video Memory.\n";
#endif
			return;
		}
	}

	int pageCount = (initialSize >> 1) / mPageSize;
	mPages.resize(pageCount);

	


	
	
}

CUMemoryPool::~CUMemoryPool()
{
	hipFree(gGlobalMemory);
	mPages.clear();
}

void CUMemoryPool::moveDevice(const Page& page)
{

	
}

void CUMemoryPool::moveHost(void* vtr, size_t size)
{
}
