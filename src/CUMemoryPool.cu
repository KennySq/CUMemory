#include"CUMemoryPool.cuh"
#include<Windows.h>

__device__ void* gGlobalMemory0 = nullptr;
__device__ void* gGlobalMemory1 = nullptr;
__device__ void* gGlobalMemory2 = nullptr;
__device__ void* gGlobalMemory3 = nullptr;

CUMemoryPool::CUMemoryPool(unsigned long long initialSize)
	: mPageSize(LowerUnit(4)), mOffset(0), mLogicalOffset(0)
	// mPageSize == 4KB == 4096 bytes
{
	if (initialSize == 0)
	{
		initialSize = 2147483648;
	}

	hipError_t error = hipMalloc(reinterpret_cast<void**>(gGlobalMemory0), initialSize >> 1);
	if (error != NULL)
	{
		initialSize = initialSize >> 1;

		error = hipMalloc(reinterpret_cast<void**>(gGlobalMemory0), initialSize >> 1);
	
		if (error != NULL)
		{
#ifdef _DEBUG
			std::cout << "Insufficient Video Memory.\n";
#endif
			return;
		}
	}

	int pageCount = (initialSize >> 1) / mPageSize;
	mPages.resize(pageCount);


	
	
}

CUMemoryPool::~CUMemoryPool()
{
	hipFree(gGlobalMemory0);
	mPages.clear();
}

CUMemoryPool::Page& CUMemoryPool::findUnfilledPage(size_t size)
{
	for (auto i : mPageMap)
	{
		if (i.second.Size >= mPageSize)
		{
			continue;
		}

		if ((mPageSize - i.second.Size) >= size)
		{
			return i.second;
		}
	}

	mPageMap.insert_or_assign(mLogicalOffset, Page());

	Page& out = mPageMap[mLogicalOffset];

	mLogicalOffset += mPageSize;

	return out;
}


