#include"pch.h"
#include"CUMemory.cuh"

__device__ void* gGlobalMemory = nullptr;

CUMemory::CUMemory(size_t initSize)
	: mSize(initSize), mOffset(0)
{
	hipError_t error = hipMalloc(reinterpret_cast<void**>(&gGlobalMemory), initSize);

	if (error != NULL)
	{
		std::cout << "Failed to init CUMemory instance.\n";
	}

	mVirtual = gGlobalMemory;

	return;
}

CUMemory::~CUMemory()
{
	hipFree(gGlobalMemory);
}

void* CUMemory::Alloc(size_t size)
{
	size_t ptr = reinterpret_cast<size_t>(mVirtual) + mOffset;

	Block block;
	block.Offset = mOffset;
	block.Size = size;

	mOffset += size;

	void* casted = reinterpret_cast<void*>(ptr);

	mBlocks.insert(std::pair<void*, Block>(casted, block));
	
	hipMemset(casted, 0xCD, size);

	return casted;
}

void CUMemory::Release(void* ptr)
{
	Block& block = mBlocks[ptr];
	size_t size = block.Size;

	mReleasedBlocks.insert(std::pair<void*, Block>(ptr, block));
	mBlocks.erase(ptr);

	
	
}
