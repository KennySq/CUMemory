#include"pch.h"
#include"CUMemory.cuh"

__device__ void* gGlobalMemory = nullptr;

CUMemory::CUMemory(size_t initSize)
	: mSize(initSize), mOffset(0)
{
	hipError_t error = hipMalloc(reinterpret_cast<void**>(&gGlobalMemory), initSize);

	if (error != NULL)
	{
		std::cout << "Failed to init CUMemory instance.\n";
	}

	mVirtual = gGlobalMemory;

	return;
}

CUMemory::~CUMemory()
{
	hipFree(gGlobalMemory);
}

void* CUMemory::Alloc(size_t size)
{
	for (std::map<void*, Block>::iterator itr = mReleasedBlocks.begin(); itr != mReleasedBlocks.end(); itr++)
	{
		if (itr->second.Size > size)
		{
			Block newBlock;
			newBlock.Offset = itr->second.Offset;
			newBlock.Size = size;
			
			if (itr->second.Size - size != 0)
			{
				Block rest;
				rest.Offset = itr->second.Offset + size;
				rest.Size = itr->second.Size - size;


				size_t restPtr = (size_t)itr->first + size;
				mReleasedBlocks.insert(std::make_pair(reinterpret_cast<void*>(restPtr), rest));
			}

			void* ptr = reinterpret_cast<void*>(reinterpret_cast<size_t>(mVirtual) + itr->second.Offset);
			mReleasedBlocks.erase(itr->first);
			mBlocks.insert(std::make_pair(ptr, newBlock));

			hipMemset(ptr, 0xCD, size);

			return ptr;
		}
	}

	size_t ptr = reinterpret_cast<size_t>(mVirtual) + mOffset;

	Block block;
	block.Offset = mOffset;
	block.Size = size;

	mOffset += size;

	void* casted = reinterpret_cast<void*>(ptr);

	mBlocks.insert(std::pair<void*, Block>(casted, block));
	
	hipMemset(casted, 0xCD, size);

	return casted;
}

void CUMemory::Release(void* ptr)
{
	Block& block = mBlocks[ptr];
	size_t size = block.Size;

	mReleasedBlocks.insert(std::pair<void*, Block>(ptr, block));
	mBlocks.erase(ptr);

	
	
}
