#include "hip/hip_runtime.h"
#include"CUMemory.cuh"
#include<iostream>
#include<Windows.h>

__global__ void memoryChecker()
{

	return;
}

template<typename _Ty>
void CopyDeivceToHost(void* vtr, _Ty* out, unsigned int count)
{
	hipMemcpy(out, vtr, sizeof(_Ty) * count, hipMemcpyDeviceToHost);
}

template<typename _Ty>
void WriteMemory(void* vtr, _Ty data)
{
	hipMemcpy(vtr, &data, sizeof(_Ty), hipMemcpyHostToDevice);
}

int main()
{
	CUMemory mem(1024);
	
	void* ptr0 = mem.Alloc(4);
	void* ptr1 = mem.Alloc(16);
	mem.Release(ptr0);
	void* ptr2 = mem.Alloc(2);
	return 0;
}