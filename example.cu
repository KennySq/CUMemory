#include "hip/hip_runtime.h"
#include"CUMemory.cuh"
#include<iostream>
#include<Windows.h>
__global__ void memoryChecker()
{

	return;
}

template<typename _Ty>
void CopyDeivceToHost(void* vtr, _Ty* out, unsigned int count)
{
	hipMemcpy(out, vtr, sizeof(_Ty) * count, hipMemcpyDeviceToHost);
}

template<typename _Ty>
void WriteMemory(void* vtr, _Ty data)
{
	hipMemcpy(vtr, &data, sizeof(_Ty), hipMemcpyHostToDevice);
}

int main()
{
	void* ptr = malloc(4);

	int* intPtr = reinterpret_cast<int*>(ptr);

	free(ptr);
	
	CUMemory memory(1024);

	CUMemory memory2(512);
	
	void* devicePtr0 = memory.Alloc(4);
	void* devicePtr1 = memory.Alloc(16);
	void* devicePtr2 = memory.Alloc(32);
	void* devicePtr3 = memory.Alloc(8);
	void* devicePtr4 = memory.Alloc(16);
	void* devicePtr5 = memory.Alloc(4);

	void* devicePtr6 = memory2.Alloc(32);

	void* devicePtr7 = memory.Alloc(4);

	int src = 17;
	int data;

	WriteMemory<int>(devicePtr7, src);

	CopyDeivceToHost<int>(devicePtr7, &data, 1);

	memoryChecker << <1, 1 >> > ();

	memory.Release(devicePtr3);

	return 0;
}