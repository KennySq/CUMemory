#include "hip/hip_runtime.h"
#include"CUMemory.cuh"
#include"CUMemoryPool.cuh"
#include"cushared_ptr.cuh"

#include<iostream>
#include<Windows.h>
#include<memory>
__global__ void memoryChecker()
{

	return;
}

template<typename _Ty>
void CopyDeivceToHost(void* vtr, _Ty* out, unsigned int count)
{
	hipMemcpy(out, vtr, sizeof(_Ty) * count, hipMemcpyDeviceToHost);
}

template<typename _Ty>
void WriteMemory(void* vtr, _Ty data)
{
	hipMemcpy(vtr, &data, sizeof(_Ty), hipMemcpyHostToDevice);
}

struct Buffer
{
	float position[4];
};

void memoryScope()
{
	cushared_ptr<int> sharedSample0 = make_cushared<int>();
	cushared_ptr<int> sharedSample1 = sharedSample0;

	return;
}

int main()
{
	void* ptr = malloc(4);

	int* intPtr = reinterpret_cast<int*>(ptr);

	free(ptr);
	
	CUMemory memory(1024);

	CUMemory memory2(512);
	CUMemoryPool pool;

	memoryScope();

	std::shared_ptr<int> a = std::shared_ptr<int>();

	make_cushared<int>();
	pool.Alloc<Buffer>(1280 * 720);


	void* devicePtr0 = memory.Alloc(4);
	void* devicePtr1 = memory.Alloc(16);
	void* devicePtr2 = memory.Alloc(32);
	void* devicePtr3 = memory.Alloc(8);
	void* devicePtr4 = memory.Alloc(16);
	void* devicePtr5 = memory.Alloc(4);

	void* devicePtr6 = memory2.Alloc(32);

	void* devicePtr7 = memory.Alloc(4);

	int src = 17;
	int data;

	WriteMemory<int>(devicePtr7, src);

	CopyDeivceToHost<int>(devicePtr7, &data, 1);

	memoryChecker << <1, 1 >> > ();

	memory.Release(devicePtr3);

	return 0;
}